#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <cmath>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace {
template<typename scalar_t>
__global__ void create_texture_image_cuda_kernel(
        const scalar_t* __restrict__ vertices_all,
        const scalar_t* __restrict__ textures,
        scalar_t* __restrict__ image,
        size_t image_size,
        size_t num_faces,
        size_t texture_size_in,
        size_t texture_size_out,
        size_t tile_width,
        scalar_t eps) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= image_size / 3) {
        return;
    }
    const int x = i % (tile_width * texture_size_out);
    const int y = i / (tile_width * texture_size_out);
    const int row = x / texture_size_out;
    const int column = y / texture_size_out;
    const int fn = row + column * tile_width;
    const int tsi = texture_size_in;

    const scalar_t* texture = &textures[fn * tsi * tsi * tsi * 3];
    const scalar_t* vertices = &vertices_all[fn * 3 * 2];
    const scalar_t* p0 = &vertices[2 * 0];
    const scalar_t* p1 = &vertices[2 * 1];
    const scalar_t* p2 = &vertices[2 * 2];

    /* */
    // if ((y % ${texture_size_out}) < (x % ${texture_size_out})) continue;

    /* compute face_inv */
    scalar_t face_inv[9] = {
        p1[1] - p2[1], p2[0] - p1[0], p1[0] * p2[1] - p2[0] * p1[1],
        p2[1] - p0[1], p0[0] - p2[0], p2[0] * p0[1] - p0[0] * p2[1],
        p0[1] - p1[1], p1[0] - p0[0], p0[0] * p1[1] - p1[0] * p0[1]};
    scalar_t face_inv_denominator = (
        p2[0] * (p0[1] - p1[1]) +
        p0[0] * (p1[1] - p2[1]) +
        p1[0] * (p2[1] - p0[1]));
    for (int k = 0; k < 9; k++) face_inv[k] /= face_inv_denominator;

    /* compute w = face_inv * p */
    scalar_t weight[3];
    scalar_t weight_sum = 0;
    for (int k = 0; k < 3; k++) {
        weight[k] = face_inv[3 * k + 0] * x + face_inv[3 * k + 1] * y + face_inv[3 * k + 2];
        weight_sum += weight[k];
    }
    for (int k = 0; k < 3; k++)
        weight[k] /= (weight_sum + eps);

    /* get texture index (scalar_t) */
    scalar_t texture_index_scalar_t[3];
    for (int k = 0; k < 3; k++) {
        scalar_t tif = weight[k] * (tsi - 1);
        tif = max(tif, 0.);
        tif = min(tif, tsi - 1 - eps);
        texture_index_scalar_t[k] = tif;
    }

    /* blend */
    scalar_t new_pixel[3] = {0, 0, 0};
    for (int pn = 0; pn < 8; pn++) {
        scalar_t w = 1;                         // weight
        int texture_index_int[3];            // index in source (int)
        for (int k = 0; k < 3; k++) {
            if ((pn >> k) % 2 == 0) {
                w *= 1 - (texture_index_scalar_t[k] - (int)texture_index_scalar_t[k]);
                texture_index_int[k] = (int)texture_index_scalar_t[k];
            }
            else {
                w *= texture_index_scalar_t[k] - (int)texture_index_scalar_t[k];
                texture_index_int[k] = (int)texture_index_scalar_t[k] + 1;
            }
        }
        int isc = texture_index_int[0] * tsi * tsi + texture_index_int[1] * tsi + texture_index_int[2];
        for (int k = 0; k < 3; k++)
            new_pixel[k] += w * texture[isc * 3 + k];
    }
    for (int k = 0; k < 3; k++)
        image[i * 3 + k] = new_pixel[k];
}

// didn't really look to see if we fuse the 2 kernels
// probably not because of synchronization issues
template<typename scalar_t>
__global__ void create_texture_image_boundary_cuda_kernel(
        scalar_t* image,
        size_t image_size,
        size_t texture_size_out,
        size_t tile_width) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= image_size / 3) {
        return;
    }

    const int x = i % (tile_width * texture_size_out);
    const int y = i / (tile_width * texture_size_out);
    if ((y % texture_size_out + 1) == (x % texture_size_out)) {
      for (int k = 0; k < 3; k++)
          image[i * 3 + k] = 
              image[ (y * tile_width * texture_size_out + (x - 1))  * 3 + k];
    }
}
}

at::Tensor create_texture_image_cuda(
        at::Tensor vertices_all,
        at::Tensor textures,
        at::Tensor image,
        float eps) {

    const auto num_faces = textures.size(0);
    const auto texture_size_in = textures.size(1);
    const auto tile_width = int(sqrt(num_faces - 1)) + 1;
    const auto texture_size_out = image.size(1) / tile_width;

    const int threads = 128;
    const int image_size = image.numel();
    const dim3 blocks ((image_size / 3 - 1) / threads + 1, 1, 1);

    AT_DISPATCH_FLOATING_TYPES(image.type(), "create_texture_image_cuda", ([&] {
      create_texture_image_cuda_kernel<scalar_t><<<blocks, threads>>>(
          vertices_all.data<scalar_t>(),
          textures.data<scalar_t>(),
          image.data<scalar_t>(),
          image_size,
          num_faces,
          texture_size_in,
          texture_size_out,
          tile_width,
          (scalar_t) eps);
      }));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error in create_texture_image: %s\n", hipGetErrorString(err));

    AT_DISPATCH_FLOATING_TYPES(image.type(), "create_texture_image_boundary", ([&] {
      create_texture_image_boundary_cuda_kernel<scalar_t><<<blocks, threads>>>(
          image.data<scalar_t>(),
          image_size,
          texture_size_out,
          tile_width);
      }));

    err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error in create_texture_image_boundary: %s\n", hipGetErrorString(err));

    return image;
}
